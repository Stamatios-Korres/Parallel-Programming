#include "hip/hip_runtime.h"
/*
 *  dmv_gpu.cu -- Template for DMV GPU kernels
 *
 *  Copyright (C) 2010-2013, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2010-2013, Vasileios Karakasis
 */ 
#include <stdio.h>
#include "dmv.h"

/*
 *  Utility function to get the thread ID within the
 *  global working space.
 */ 
__device__ int get_global_tid()
{
	return (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x  *blockDim.y + blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Utility function to get the thread ID within the
 *  local/block working space.
 */ 
__device__ int get_local_tid()
{
	return blockDim.x*threadIdx.y + threadIdx.x;
	
	
}

__global__ void dmv_gpu_naive(const value_t *a, const value_t *x, value_t *y, size_t n)
{
	value_t sum=0;
	int j;
	int row = blockIdx.x*blockDim.x+threadIdx.x;
	for(j=0;j<n;j++){
     		sum += a[row*n+j]*x[j];
  	}
   	y[row]=sum;
}

__global__ void dmv_gpu_coalesced(const value_t *a, const value_t *x,
                                  value_t *y, size_t n)
{
	int row = blockIdx.x*blockDim.x+threadIdx.x;
        value_t sum=0;
        int j;
        for(j=0;j<n;j++){
                sum += a[row+j*n]*x[j];
        }
        y[row]=sum;
}

__global__ void dmv_gpu_shmem(const value_t *a, const value_t *x, value_t *y,size_t n)
{
	extern __shared__ value_t sharing[];
	int global = blockIdx.x*blockDim.x+threadIdx.x;
	int i,j;
	value_t sum = 0;
	for(i=0;i<gridDim.x;i++){
		__syncthreads();
		sharing[threadIdx.x] = x[threadIdx.x + i*blockDim.x];
		__syncthreads();
		for(j=0;j<blockDim.x;j++){
				sum = sum + (a[ global + j*n + (blockDim.x*n)*i ] * sharing[j]);
		}
	}
        y[global]=sum;
}
